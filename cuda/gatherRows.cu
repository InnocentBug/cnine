#include "hip/hip_runtime.h"
/*
 * This file is part of cnine, a lightweight C++ tensor library. 
 *  
 * Copyright (c) 2021, Imre Risi Kondor
 *
 * This source code file is subject to the terms of the noncommercial 
 * license distributed with cnine in the file LICENSE.TXT. Commercial 
 * use is prohibited. All redistributed versions of this file (in 
 * original or modified form) must retain this copyright notice and 
 * must be accompanied by a verbatim copy of the license. 
 *
 */

#ifndef _gatherRows_cu
#define _gatherRows_cu

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include "Cnine_base.hpp"
#include "Rtensor2_view.hpp"
#include "GatherMapB.hpp"
#include "WeightedGatherMapB.hpp"


__global__ void gatherRows_kernel(float* rarr, const int rs0, const float* xarr, const int xs0, const int* ix, const int N, const int nc){
  extern __shared__ unsigned char _shared[]; 
  
  int i=blockIdx.x*blockDim.x+threadIdx.x;
  if(i>=N) return;

  int t=threadIdx.y;
  if(t>=nc) return;

  const int* row=ix+2*N+ix[2*i];
  int n=ix[2*i+1]-1;
  int target=row[0];

  float a=0;
  for(int j=0; j<n; j++)
    a+=xarr[row[j+1]*xs0+t];
  rarr[target*rs0+t]+=a;

}


__global__ void gatherRowsw_kernel(float* rarr, const int rs0, const float* xarr, const int xs0, const int* ix, const int N, const int nc){
  extern __shared__ unsigned char _shared[]; 
  
  int i=blockIdx.x*blockDim.x+threadIdx.x;
  if(i>=N) return;

  int t=threadIdx.y;
  if(t>=nc) return;

  const int* row=ix+2*N+ix[2*i];
  int n=(ix[2*i+1]-1)/2;
  int target=row[0];

  float a=0;
  for(int j=0; j<n; j++)
    a+=xarr[row[2*j+1]*xs0+t]*row[2*j+2];
  rarr[target*rs0+t]+=a;

}


__global__ void gatherRows_kernel(float* rarr, const int rs0, const float* xarr, const int xs0, const int* g, const int N, const int K, const int nc){

  int i=blockIdx.x*blockDim.x+threadIdx.x;
  if(i>=N) return;

  int t=threadIdx.y;
  if(t>=nc) return;

  const int* row=g+(K+1)*i;
  int target=row[0];

  float a=0;
  for(int j=0; j<K; j++)
    a+=xarr[row[j+1]*xs0+t];
  rarr[target*rs0+t]+=a;

}


namespace cnine{

  void gatherRows_cu(const Rtensor2_view& r, const Rtensor2_view& x, const GatherMapB& g, const hipStream_t& stream){
    int nc=x.n1;
    CNINE_ASSRT(r.dev==1);
    CNINE_ASSRT(x.dev==1);
    CNINE_ASSRT(nc<=1024);
    CNINE_ASSRT(x.s1==1);
    CNINE_ASSRT(r.s1==1);

    int nwarps=roundup(nc,32)/32;
    int multi=32/nwarps;
    multi=1; // muti seems to make things worse!
    dim3 threads(multi,nwarps*32);
    //cout<<multi<<" "<<nwarps<<" "<<g.size()<<" "<<(g.size()-1)/multi+1<<endl;

    gatherRows_kernel<<<(g.size()-1)/multi+1,threads,0,stream>>>
      (r.arr,r.s0,x.arr,x.s0,g.get_arrg(1),g.size(),nc);
    hipDeviceSynchronize();
  }


  void gatherRowsw_cu(const Rtensor2_view& r, const Rtensor2_view& x, const WeightedGatherMapB& g, const hipStream_t& stream){
    int nc=r.n1;
    CNINE_ASSRT(r.dev==1);
    CNINE_ASSRT(x.dev==1);
    CNINE_ASSRT(x.n1==nc);
    CNINE_ASSRT(nc<=1024);
    CNINE_ASSRT(x.s1==1);
    CNINE_ASSRT(r.s1==1);

    int nwarps=roundup(nc,32)/32;
    int multi=32/nwarps;
    multi=1; // muti seems to make things worse!
    dim3 threads(multi,nwarps*32);
    //cout<<multi<<" "<<nwarps<<" "<<g.size()<<" "<<(g.size()-1)/multi+1<<endl;
    //cout<<g.arr.dir<<endl;

    gatherRowsw_kernel<<<(g.size()-1)/multi+1,threads,0,stream>>>
      (r.arr,r.s0,x.arr,x.s0,g.get_arrg(1),g.size(),nc);
    hipDeviceSynchronize();
  }


  void gatherRows_cu(const Rtensor2_view& r, const Rtensor2_view& x, const FixedkGatherMap& g, const hipStream_t& stream){
    int nc=r.n1;
    CNINE_ASSRT(r.dev==1);
    CNINE_ASSRT(x.dev==1);
    CNINE_ASSRT(x.n1==nc);
    CNINE_ASSRT(nc<=1024);
    CNINE_ASSRT(x.s1==1);
    CNINE_ASSRT(r.s1==1);
    CNINE_ASSRT(g.strides(0)==g.getk()+1);
    CNINE_ASSRT(g.strides(1)==1);

    int nwarps=roundup(nc,32)/32;
    int multi=32/nwarps;
    dim3 threads(multi,nwarps*32);

    gatherRows_kernel<<<(g.size()-1)/multi+1,threads,0,stream>>> // changed
      (r.arr,r.s0,x.arr,x.s0,g.arr.ptr(),g.size(),g.getk(),nc);
  }

}

#endif 
